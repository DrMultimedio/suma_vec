
#include "hip/hip_runtime.h"

#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b)
{

}

void suma_vectores
(
	float *pA,
	float *pB,
	float *pC,
	const int & crNumElements
)
{
	for (int i = 0; i < crNumElements; ++i)
	{
		pC[i] = pA[i] + pB[i];
	}
}
__global__

void kernel_suma_vectores
(
	const float* cpA,
	const float* cpB,
	float* pC,
	const int cNumElements
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	pC[idx] = cpA[idx] + cpB[idx];
}
int main()
{
    //paso 1 -> Inicializaci�n
	hipSetDevice(0); //Esta funci�n le dice al framework: voy a usar la tarjeta x

	//paso 2 -> Declaraci�n y reserva
	const int kNumElements = 25600; //numero al azar
	size_t kNumBytes = kNumElements * sizeof(float); //bytes totales para reservar y pasarselo al malloc
	//declaro los vectores en la CPU (HOST)
	float *h_A_ = (float *)malloc(kNumBytes);
	float *h_B_ = (float *)malloc(kNumBytes);
	float *h_C_ = (float *)malloc(kNumBytes);

	if (h_A_ == NULL || h_B_ == NULL || h_C_ == NULL) {
		std::cerr << "La memoria ha fallado lol \n";
		getchar();
		exit(-1);
	}
	float* d_A_ = NULL;
	float* d_B_ = NULL;
	float* d_C_ = NULL;

	//declaro los vectores en la GPU (HOST)
	hipMalloc((void **)&d_A_, kNumBytes);
	hipMalloc((void **)&d_B_, kNumBytes);
	hipMalloc((void **)&d_C_, kNumBytes);

	//le ponemos numeros aleatorios en lugar de los que les apetezca al cacharro

	for (int i = 0; i < kNumElements; ++i)
	{
		h_A_[i] = rand() / RAND_MAX;
		h_B_[i] = rand() / RAND_MAX;
 	}
	
	//PASO 3: Transferencia CPU a la GPU

	hipMemcpy(d_A_, h_A_, kNumBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B_, h_B_, kNumBytes, hipMemcpyHostToDevice);

	//PASO 4: Ejecuci�n de le kernel

	const int threads_per_block_ = 256;
	const int blocks_per_grid_ = kNumElements / threads_per_block_;

	dim3 block(threads_per_block_, 1, 1);
	dim3 grid(blocks_per_grid_, 1, 1);

	kernel_suma_vectores<<<grid, block >>>(d_A_, d_B_, d_C_, kNumElements);
	hipError_t err_ = hipGetLastError();
	if (err_ != hipSuccess)
	{
		std::cerr << hipGetErrorString(err_) << "\n";
		getchar();
		exit(-1);
	}

	//paso 5: transferencia de la gpu a la cpu

	hipMemcpy(h_C_, d_C_, kNumBytes, hipMemcpyDeviceToHost);

	//paso 6: Comprobaci�n y liberaci�n

	for (int i = 0; i < kNumElements; i++) {
		if (fabs(h_A_[i] + h_B_[i] - h_C_[i]) > 1e-5) {
			std::cerr << "fallo de verificaci�n en la posicion" << i << "\n";
		}
	}

	free(h_A_);
	free(h_B_);
	free(h_C_);
	hipFree(d_A_);
	hipFree(d_B_);
	hipFree(d_C_);

	hipDeviceReset();
	std::cout << "Test Passed \n";
	getchar();
}
